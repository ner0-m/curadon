#include "hip/hip_runtime.h"
#include "doctest/doctest.h"

#include <cmath>
#include <iomanip>
#include <sstream>
#include <type_traits>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "curadon/backprojection.hpp"
#include "curadon/bmp.hpp"

#define gpuErrchk(answer)                                                                          \
    { gpuAssert((answer), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: '%s: %s' (%d) in %s:%d\n", hipGetErrorName(code),
                hipGetErrorString(code), code, file, line);
        if (abort) {
            exit(code);
        }
    }
}

void draw(float *data, int slice, int width, int height, int depth);

curad::geometry setup_geom(size_t volsize, size_t width, size_t height, float angle, float DSO,
                           float DSD) {
    curad::geometry geo;
    geo.DSO = DSO;
    geo.DSD = DSD;

    geo.det_shape_ = curad::Vec<std::uint64_t, 2>{width, height};
    geo.det_spacing_ = curad::Vec<float, 2>{1, 1};
    geo.det_size_ = geo.det_shape_ * geo.det_spacing_;
    geo.det_offset_ = curad::Vec<float, 2>{0, 0};
    geo.det_rotation_ = curad::Vec<float, 3>{0, 0, 0};
    geo.COR_ = 0;

    geo.vol_shape_ = curad::Vec<std::uint64_t, 3>{volsize, volsize, volsize};
    geo.vol_spacing_ = curad::Vec<float, 3>{1, 1, 1};
    geo.vol_size_ = geo.vol_shape_ * geo.vol_spacing_;

    geo.phi_ = angle * M_PI / 180;
    geo.theta_ = 0;
    geo.psi_ = 0;

    return geo;
}

TEST_CASE("test_backprojection") {

    const auto volsize = 64;
    auto [data, width, height, nangles, angles, DSO, DSD] = curad::easy::read("demofile2.txt");
    // std::cout << "Width: " << width << " Height: " << height << " nangles: " << nangles << "\n";

    CHECK_EQ(width, 16);
    CHECK_EQ(height, 16);
    CHECK_EQ(nangles, 1);
    CHECK_EQ(DSO, 100);
    CHECK_EQ(DSD, 120);

    thrust::host_vector<float> host_sino(width * height * nangles, 0);
    std::copy(data.begin(), data.end(), host_sino.begin());

    thrust::device_vector<float> sino = host_sino;
    auto sino_ptr = thrust::raw_pointer_cast(sino.data());

    thrust::device_vector<float> volume(volsize * volsize * volsize, 0);
    auto volume_ptr = thrust::raw_pointer_cast(volume.data());
    gpuErrchk(hipDeviceSynchronize());

    auto det_shape = curad::Vec<std::uint64_t, 2>{width, height};
    auto vol_shape = curad::Vec<std::uint64_t, 3>{volsize, volsize, volsize};

    // auto vol_spacing = curad::Vec<float, 3>{1, 1, 1};
    auto vol_spacing = curad::Vec<float, 3>{3, 3, 3};
    auto vol_size = vol_shape * vol_spacing;
    auto vol_offset = curad::Vec<float, 3>{0, 0, 0};

    curad::Vec<float, 3> source({0, 0, -DSO});

    const auto stride_x = 1;
    const auto stride_y = volsize;
    const auto stride_z = volsize * volsize;

    // allocate cuda array for sinogram
    const hipExtent extent_alloc = make_hipExtent(width, height, curad::num_projects_per_kernel);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array_cu;
    hipMalloc3DArray(&array_cu, &channelDesc, extent_alloc);
    gpuErrchk(hipPeekAtLastError());

    hipTextureObject_t tex;

    auto num_kernel_calls =
        (nangles + curad::num_projects_per_kernel - 1) / curad::num_projects_per_kernel;
    for (int i = 0; i < num_kernel_calls; ++i) {
        auto proj_idx = i * curad::num_projects_per_kernel;

        // Copy to cuda array
        hipMemcpy3DParms copyParams = {0};
        gpuErrchk(hipPeekAtLastError());

        auto ptr = sino_ptr + proj_idx * width * height;
        copyParams.srcPtr = make_hipPitchedPtr((void *)ptr, width * sizeof(float), width, height);

        auto projections_left = nangles - (i * curad::num_projects_per_kernel);
        const hipExtent extent = make_hipExtent(
            width, height, std::min<int>(curad::num_projects_per_kernel, projections_left));
        // std::cout << "sino size: " << sino.size() << "\n";
        // std::cout << "offset: " << proj_idx * width * height << "\n";
        // std::cout << "extent: " << extent.width << " " << extent.height << " " << extent.depth
        //           << "\n";
        gpuErrchk(hipPeekAtLastError());
        copyParams.dstArray = array_cu;
        copyParams.extent = extent;
        copyParams.kind = hipMemcpyDefault;
        hipMemcpy3DAsync(&copyParams, 0); // TODO: use stream pool
        hipStreamSynchronize(0);
        gpuErrchk(hipPeekAtLastError());

        hipResourceDesc texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = array_cu;
        hipTextureDesc texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;

        hipCreateTextureObject(&tex, &texRes, &texDescr, NULL);
        gpuErrchk(hipPeekAtLastError());

        std::vector<curad::Vec<float, 3>> vol_origins;
        std::vector<curad::Vec<float, 3>> delta_xs;
        std::vector<curad::Vec<float, 3>> delta_ys;
        std::vector<curad::Vec<float, 3>> delta_zs;
        for (int j = 0; j < curad::num_projects_per_kernel; ++j) {
            float angle = angles[proj_idx + j] * M_PI / 180.f;

            curad::Vec<float, 3> init_vol_origin = -vol_size / 2.f + vol_spacing / 2.f + vol_offset;
            auto vol_origin = curad::detail::rotate_yzy(init_vol_origin, angle, 0.f, 0.f);
            vol_origins.push_back(vol_origin);

            curad::Vec<float, 3> init_delta;
            init_delta = init_vol_origin;
            init_delta[0] += vol_spacing[0];
            init_delta = curad::detail::rotate_yzy(init_delta, angle, 0.f, 0.f);
            delta_xs.push_back(init_delta - vol_origin);

            init_delta = init_vol_origin;
            init_delta[1] += vol_spacing[1];
            init_delta = curad::detail::rotate_yzy(init_delta, angle, 0.f, 0.f);
            delta_ys.push_back(init_delta - vol_origin);

            init_delta = init_vol_origin;
            init_delta[2] += vol_spacing[2];
            init_delta = curad::detail::rotate_yzy(init_delta, angle, 0.f, 0.f);
            delta_zs.push_back(init_delta - vol_origin);
        }

        hipMemcpyToSymbol(HIP_SYMBOL(curad::dev_vol_origin), vol_origins.data(),
                           sizeof(curad::Vec<float, 3>) * curad::num_projects_per_kernel, 0,
                           hipMemcpyDefault);
        hipMemcpyToSymbol(HIP_SYMBOL(curad::dev_delta_x), delta_xs.data(),
                           sizeof(curad::Vec<float, 3>) * curad::num_projects_per_kernel, 0,
                           hipMemcpyDefault);
        hipMemcpyToSymbol(HIP_SYMBOL(curad::dev_delta_y), delta_ys.data(),
                           sizeof(curad::Vec<float, 3>) * curad::num_projects_per_kernel, 0,
                           hipMemcpyDefault);
        hipMemcpyToSymbol(HIP_SYMBOL(curad::dev_delta_z), delta_zs.data(),
                           sizeof(curad::Vec<float, 3>) * curad::num_projects_per_kernel, 0,
                           hipMemcpyDefault);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        int divx = 16;
        int divy = 32;
        int divz = curad::num_voxels_per_thread;

        dim3 threads_per_block(divx, divy, 1);

        int block_x = (vol_shape[0] + divx - 1) / divx;
        int block_y = (vol_shape[1] + divy - 1) / divy;
        int block_z = (vol_shape[2] + divz - 1) / divz;
        dim3 num_blocks(block_x, block_y, block_z);
        curad::kernel_backprojection_single<<<num_blocks, threads_per_block>>>(
            volume_ptr, stride_x, stride_y, stride_z, source, vol_shape, DSD, DSO, det_shape, i,
            nangles, tex);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    hipDestroyTextureObject(tex);

    thrust::host_vector<float> vol_host = volume;

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    std::transform(vol_host.begin(), vol_host.end(), vol_host.begin(),
                   [&](auto x) { return x / 50; });

    // std::cout << "max of vol_host: " << *std::max_element(vol_host.begin(), vol_host.end())
    //           << std::endl;
    //
    // std::cout << "max of data: " << *std::max_element(data.begin(), data.end()) << std::endl;

    // auto max_elem = *std::max_element(data.begin(), data.end());
    auto max_elem = *std::max_element(vol_host.begin(), vol_host.end());
    std::transform(vol_host.begin(), vol_host.end(), vol_host.begin(),
                   [&](auto x) { return x / max_elem; });
    const auto slice = volsize / 2;
    draw(thrust::raw_pointer_cast(vol_host.data()), slice, volsize, volsize, volsize);
    // draw(data.data(), slice, width, height, nangles);
}

// Include GLEW. Always include it before gl.h and glfw3.h, since it's a bit magic.
#include <GL/glew.h>

#include <GL/glut.h>
#include <GLFW/glfw3.h>

void framebuffer_size_callback(GLFWwindow *window, int width, int height) {
    glViewport(0, 0, width, height);
}

void processInput(GLFWwindow *window, unsigned int &curtex, unsigned int maxtex) {
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, true);
    } else if (glfwGetKey(window, GLFW_KEY_UP) == GLFW_PRESS) {
        curtex = (curtex + 1) % maxtex;
        // std::cout << "Current texture: " << curtex << " / " << maxtex << std::endl;
    } else if (glfwGetKey(window, GLFW_KEY_DOWN) == GLFW_PRESS) {
        curtex = (curtex - 1) % maxtex;
        // std::cout << "Current texture: " << curtex << " / " << maxtex << std::endl;
    }
}

// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 600;

#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

class Shader {
  public:
    unsigned int ID;
    // constructor generates the shader on the fly
    // ------------------------------------------------------------------------
    Shader(const char *vertexPath, const char *fragmentPath) {
        // 1. retrieve the vertex/fragment source code from filePath
        std::string vertexCode;
        std::string fragmentCode;
        std::ifstream vShaderFile;
        std::ifstream fShaderFile;
        // ensure ifstream objects can throw exceptions:
        vShaderFile.exceptions(std::ifstream::failbit | std::ifstream::badbit);
        fShaderFile.exceptions(std::ifstream::failbit | std::ifstream::badbit);
        try {
            // open files
            vShaderFile.open(vertexPath);
            fShaderFile.open(fragmentPath);
            std::stringstream vShaderStream, fShaderStream;
            // read file's buffer contents into streams
            vShaderStream << vShaderFile.rdbuf();
            fShaderStream << fShaderFile.rdbuf();
            // close file handlers
            vShaderFile.close();
            fShaderFile.close();
            // convert stream into string
            vertexCode = vShaderStream.str();
            fragmentCode = fShaderStream.str();
        } catch (std::ifstream::failure &e) {
            std::cout << "ERROR::SHADER::FILE_NOT_SUCCESSFULLY_READ: " << e.what() << std::endl;
        }
        const char *vShaderCode = vertexCode.c_str();
        const char *fShaderCode = fragmentCode.c_str();
        // 2. compile shaders
        unsigned int vertex, fragment;
        // vertex shader
        vertex = glCreateShader(GL_VERTEX_SHADER);
        glShaderSource(vertex, 1, &vShaderCode, NULL);
        glCompileShader(vertex);
        checkCompileErrors(vertex, "VERTEX");
        // fragment Shader
        fragment = glCreateShader(GL_FRAGMENT_SHADER);
        glShaderSource(fragment, 1, &fShaderCode, NULL);
        glCompileShader(fragment);
        checkCompileErrors(fragment, "FRAGMENT");
        // shader Program
        ID = glCreateProgram();
        glAttachShader(ID, vertex);
        glAttachShader(ID, fragment);
        glLinkProgram(ID);
        checkCompileErrors(ID, "PROGRAM");
        // delete the shaders as they're linked into our program now and no longer necessary
        glDeleteShader(vertex);
        glDeleteShader(fragment);
    }
    // activate the shader
    // ------------------------------------------------------------------------
    void use() { glUseProgram(ID); }
    // utility uniform functions
    // ------------------------------------------------------------------------
    void setBool(const std::string &name, bool value) const {
        glUniform1i(glGetUniformLocation(ID, name.c_str()), (int)value);
    }
    // ------------------------------------------------------------------------
    void setInt(const std::string &name, int value) const {
        glUniform1i(glGetUniformLocation(ID, name.c_str()), value);
    }
    // ------------------------------------------------------------------------
    void setFloat(const std::string &name, float value) const {
        glUniform1f(glGetUniformLocation(ID, name.c_str()), value);
    }

  private:
    // utility function for checking shader compilation/linking errors.
    // ------------------------------------------------------------------------
    void checkCompileErrors(unsigned int shader, std::string type) {
        int success;
        char infoLog[1024];
        if (type != "PROGRAM") {
            glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
            if (!success) {
                glGetShaderInfoLog(shader, 1024, NULL, infoLog);
                std::cout << "ERROR::SHADER_COMPILATION_ERROR of type: " << type << "\n"
                          << infoLog
                          << "\n -- --------------------------------------------------- -- "
                          << std::endl;
            }
        } else {
            glGetProgramiv(shader, GL_LINK_STATUS, &success);
            if (!success) {
                glGetProgramInfoLog(shader, 1024, NULL, infoLog);
                std::cout << "ERROR::PROGRAM_LINKING_ERROR of type: " << type << "\n"
                          << infoLog
                          << "\n -- --------------------------------------------------- -- "
                          << std::endl;
            }
        }
    }
};

void draw(float *data, int slice, int width, int height, int depth) {
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    // glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);

    GLFWwindow *window = glfwCreateWindow(800, 600, "LearnOpenGL", NULL, NULL);
    if (window == NULL) {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return;
    }
    glfwMakeContextCurrent(window);

    glViewport(0, 0, 800, 600);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // start GLEW extension handler
    glewExperimental = GL_TRUE;
    glewInit();

    // build and compile our shader program
    // ------------------------------------
    // vertex shader
    Shader ourShader("texture.vs", "texture.fs");

    // set up vertex data (and buffer(s)) and configure vertex attributes
    // ------------------------------------------------------------------
    // clang-format off
    float vertices[] = {
        // positions          // colors           // texture coords
         1.f,  1.f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
         1.f, -1.f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
        -1.f, -1.f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
        -1.f,  1.f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    // clang-format on

    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)0);
    glEnableVertexAttribArray(0);
    // color attribute
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);
    // texture coord attribute
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(6 * sizeof(float)));
    glEnableVertexAttribArray(2);

    // load and create a texture
    // -------------------------
    std::vector<unsigned int> textures;
    textures.reserve(depth);
    for (int i = 0; i < depth; ++i) {
        // texture 1
        // ---------
        unsigned int tex;
        glGenTextures(1, &tex);
        glBindTexture(GL_TEXTURE_2D, tex);
        // set the texture wrapping parameters
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);

        // set texture filtering parameters
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

        // load image, create texture and generate mipmaps
        const auto offset = i * width * height;
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RED, width, height, 0, GL_RED, GL_FLOAT, data + offset);

        textures.push_back(tex);
    }

    // tell opengl for each sampler to which texture unit it belongs to (only has to be done
    // once)
    // -------------------------------------------------------------------------------------------
    ourShader.use(); // don't forget to activate/use the shader before setting uniforms!
    // or set it via the texture class
    ourShader.setInt("texture1", 0);

    unsigned int curtex = 0;
    while (!glfwWindowShouldClose(window)) {
        processInput(window, curtex, textures.size());

        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // bind textures on corresponding texture units
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, textures[curtex]);

        // render container
        ourShader.use();
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    // optional: de-allocate all resources once they've outlived their purpose:
    // ------------------------------------------------------------------------
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);

    glfwTerminate();
}
